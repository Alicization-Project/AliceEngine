#include "hip/hip_runtime.h"
/*
 * This program uses the device CURAND API to calculate what
 * proportion of quasi-random 3D points fall within a sphere
 * of radius 1, and to derive the volume of the sphere.
 *
 * In particular it uses 64 bit scrambled Sobol direction
 * vectors from the file sobol_direction_vectors.h, to
 * generate double precision uniform samples.
 *
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#define THREADS_PER_BLOCK 64
#define BLOCK_COUNT 64
#define TOTAL_THREADS (THREADS_PER_BLOCK * BLOCK_COUNT)

 /* Number of 64-bit vectors per dimension */
#define VECTOR_SIZE 64


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

/* This kernel initializes state per thread for each of x, y, and z */

__global__ void setup_kernel(unsigned long long * sobolDirectionVectors,
	unsigned long long *sobolScrambleConstants,
	hiprandStateScrambledSobol64 *state)
{
	int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	int dim = 3 * id;
	/* Each thread uses 3 different dimensions */
	hiprand_init(sobolDirectionVectors + VECTOR_SIZE * dim,
		sobolScrambleConstants[dim],
		1234,
		&state[dim]);

	hiprand_init(sobolDirectionVectors + VECTOR_SIZE * (dim + 1),
		sobolScrambleConstants[dim + 1],
		1234,
		&state[dim + 1]);

	hiprand_init(sobolDirectionVectors + VECTOR_SIZE * (dim + 2),
		sobolScrambleConstants[dim + 2],
		1234,
		&state[dim + 2]);
}

/* This kernel generates random 3D points and increments a counter if
 * a point is within a unit sphere
 */
__global__ void generate_kernel(hiprandStateScrambledSobol64 *state,
	int n,
	long long int *result)
{
	int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	int baseDim = 3 * id;
	long long int count = 0;
	double x, y, z;

	/* Generate quasi-random double precision coordinates */
	for (int i = 0; i < n; i++) {
		x = hiprand_uniform_double(&state[baseDim]);
		y = hiprand_uniform_double(&state[baseDim + 1]);
		z = hiprand_uniform_double(&state[baseDim + 2]);

		/* Check if within sphere of radius 1 */
		if ((x*x + y * y + z * z) < 1.0) {
			count++;
		}
	}
	/* Store results */
	result[id] += count;
}

int main(int argc, char *argv[])
{
	int i;
	long long total;
	hiprandStateScrambledSobol64 *devSobol64States;
	hiprandDirectionVectors64_t *hostVectors64;
	unsigned long long int * hostScrambleConstants64;
	unsigned long long int * devDirectionVectors64;
	unsigned long long int * devScrambleConstants64;
	long long int *devResults, *hostResults;
	int sampleCount = 10000;
	int iterations = 100;
	double fraction;
	double pi = 3.1415926535897932;

	/* Allow over-ride of sample count */
	if (argc == 2) {
		sscanf(argv[1], "%d", &sampleCount);
	}

	/* Allocate space for results on host */
	hostResults = (long long int*)calloc(TOTAL_THREADS,
		sizeof(long long int));

	/* Allocate space for results on device */
	CUDA_CALL(hipMalloc((void **)&devResults,
		TOTAL_THREADS * sizeof(long long int)));

	/* Set results to 0 */
	CUDA_CALL(hipMemset(devResults, 0,
		TOTAL_THREADS * sizeof(long long int)));

	/* Get pointers to the 64 bit scrambled direction vectors and constants*/
	CURAND_CALL(hiprandGetDirectionVectors64(&hostVectors64,
		HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6));

	CURAND_CALL(hiprandGetScrambleConstants64(&hostScrambleConstants64));


	/* Allocate memory for 3 states per thread (x, y, z), each state to get a unique dimension */
	CUDA_CALL(hipMalloc((void **)&devSobol64States,
		TOTAL_THREADS * 3 * sizeof(hiprandStateScrambledSobol64)));

	/* Allocate memory and copy 3 sets of vectors per thread to the device */

	CUDA_CALL(hipMalloc((void **)&(devDirectionVectors64),
		3 * TOTAL_THREADS * VECTOR_SIZE * sizeof(long long int)));

	CUDA_CALL(hipMemcpy(devDirectionVectors64, hostVectors64,
		3 * TOTAL_THREADS * VECTOR_SIZE * sizeof(long long int),
		hipMemcpyHostToDevice));

	/* Allocate memory and copy 3 scramble constants (one costant per dimension)
	   per thread to the device */

	CUDA_CALL(hipMalloc((void **)&(devScrambleConstants64),
		3 * TOTAL_THREADS * sizeof(long long int)));

	CUDA_CALL(hipMemcpy(devScrambleConstants64, hostScrambleConstants64,
		3 * TOTAL_THREADS * sizeof(long long int),
		hipMemcpyHostToDevice));

	/* Initialize the states */

	setup_kernel << <BLOCK_COUNT, THREADS_PER_BLOCK >> > (devDirectionVectors64,
		devScrambleConstants64,
		devSobol64States);

	/* Generate and count quasi-random points  */

	for (i = 0; i < iterations; i++) {
		generate_kernel << <BLOCK_COUNT, THREADS_PER_BLOCK >> > (devSobol64States, sampleCount, devResults);
	}

	/* Copy device memory to host */

	CUDA_CALL(hipMemcpy(hostResults,
		devResults,
		TOTAL_THREADS * sizeof(long long int),
		hipMemcpyDeviceToHost));

	/* Tally and show result */

	total = 0;
	for (i = 0; i < TOTAL_THREADS; i++) {
		total += hostResults[i];
	}

	fraction = (double)total / ((double)TOTAL_THREADS * (double)sampleCount * (double)iterations);
	printf("Fraction inside sphere was %g\n", fraction);
	printf("(4/3) pi = %g, sampled volume = %g\n", (4.0*pi / 3.0), 8.0 * fraction);

	/* Cleanup */

	CUDA_CALL(hipFree(devSobol64States));
	CUDA_CALL(hipFree(devDirectionVectors64));
	CUDA_CALL(hipFree(devScrambleConstants64));
	CUDA_CALL(hipFree(devResults));
	free(hostResults);
	printf("^^^^ kernel_sobol_example PASSED\n");


	return EXIT_SUCCESS;
}